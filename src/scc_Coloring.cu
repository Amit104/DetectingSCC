#include "hip/hip_runtime.h"
#include "scc.h"
#include "scc_kernels.h"
using namespace std;


void wSlota(uint32_t CSize, uint32_t RSize, uint32_t *Fc, uint32_t *Fr, uint32_t * Bc, uint32_t * Br, bool t1, bool t2, int warpSize){

    //Set the device which exclusively used by this program
    hipSetDevice(7);

    float sccTime=0;
    hipEvent_t sccTimeStart, sccTimeStop;
    hipEventCreate(&sccTimeStart);
    hipEventCreate(&sccTimeStop);
    hipEventRecord(sccTimeStart, 0);

//-----------GPU initialization---------------------------->
    uint32_t* d_Fr = NULL;
    uint32_t* d_Br = NULL;
    uint32_t* d_Fc = NULL;
    uint32_t* d_Bc = NULL;
    uint32_t* d_pivots = NULL;

    uint32_t* d_range = NULL;
    uint8_t* d_tags = NULL;
    uint8_t* tags = new uint8_t[RSize+1];

    bool volatile* d_terminatef = NULL;
    bool terminatef = false;

    bool volatile* d_terminateb = NULL;
    bool terminateb = false;

    int FWD_iterations = 0;
    int BWD_iterations = 0;
    uint32_t iterations = 0;

    const uint32_t max_pivot_count = 1;

    hipError_t e1, e2, e3, e4, e5, e6, e7, e8, e9;
    CUDA_SAFE_CALL( e1 = hipMalloc( (void**) &d_Fc, CSize * sizeof(uint32_t) ));
    CUDA_SAFE_CALL( e2 = hipMalloc( (void**) &d_Fr, (RSize + 2) * sizeof(uint32_t) ));
    CUDA_SAFE_CALL( e3 = hipMalloc( (void**) &d_Bc, CSize * sizeof(uint32_t) ));
    CUDA_SAFE_CALL( e4 = hipMalloc( (void**) &d_Br, (RSize + 2) * sizeof(uint32_t) ));
    CUDA_SAFE_CALL( e5 = hipMalloc( (void**) &d_range,  (RSize + 1) * sizeof(uint32_t)));
    CUDA_SAFE_CALL( e6 = hipMalloc( (void**) &d_tags,  (RSize + 1) * sizeof(uint8_t)));
    CUDA_SAFE_CALL( e7 = hipMalloc( (void**) &d_pivots, max_pivot_count * sizeof(uint32_t) ));
    CUDA_SAFE_CALL( e8 = hipMalloc( (void**) &d_terminatef, sizeof(bool) ));
    CUDA_SAFE_CALL( e9 = hipMalloc( (void**) &d_terminateb, sizeof(bool) ));

    if (e1 == hipErrorOutOfMemory || e2 == hipErrorOutOfMemory ||
        e3 == hipErrorOutOfMemory || e4 == hipErrorOutOfMemory ||
        e5 == hipErrorOutOfMemory || e6 == hipErrorOutOfMemory ||
        e7 == hipErrorOutOfMemory || e8 == hipErrorOutOfMemory || e9 == hipErrorOutOfMemory) {
        throw "Error: Not enough memory on GPU\n";
    }

    CUDA_SAFE_CALL( hipMemcpy( d_Fc, Fc, CSize * sizeof(uint32_t), hipMemcpyHostToDevice ));
    CUDA_SAFE_CALL( hipMemcpy( d_Fr, Fr, (RSize + 2) * sizeof(uint32_t), hipMemcpyHostToDevice ));
    CUDA_SAFE_CALL( hipMemcpy( d_Bc, Bc, CSize * sizeof(uint32_t), hipMemcpyHostToDevice ));
    CUDA_SAFE_CALL( hipMemcpy( d_Br, Br, (RSize + 2) * sizeof(uint32_t), hipMemcpyHostToDevice ));
    
    CUDA_SAFE_CALL( hipMemset( d_range, 0, (RSize + 1) * sizeof(uint32_t)));
    CUDA_SAFE_CALL( hipMemset( d_tags, 0, (RSize + 1) * sizeof(uint8_t)));

    dim3 gridfb;
    if((RSize * warpSize + BLOCKSIZE - 1)/BLOCKSIZE > MaxXDimOfGrid) {
        int dim = ceill(sqrt(RSize * warpSize / BLOCKSIZE));
        gridfb.x = dim;
        gridfb.y = dim;
        gridfb.z = 1;
    }else{
        gridfb.x = (RSize * warpSize + BLOCKSIZE - 1)/BLOCKSIZE;
        gridfb.y = 1;
        gridfb.z = 1;
    }

    //for vertex-to-thread mapping
    dim3 grid;
    if((RSize + BLOCKSIZE - 1)/BLOCKSIZE > MaxXDimOfGrid) {
        int dim = ceill(sqrt(RSize / BLOCKSIZE));
        grid.x = dim;
        grid.y = dim;
        grid.z = 1;
    }else{
        grid.x = (RSize + BLOCKSIZE - 1)/BLOCKSIZE;
        grid.y = 1;
        grid.z = 1;
    }

    dim3 threads(BLOCKSIZE, 1, 1);

#ifdef _DEBUG
float pivotTime = 0, temp = 0, bTime = 0, pTime = 0, trim1Time = 0, updateTime = 0, bfsTime = 0;
hipEvent_t bTimeStart, bTimeStop, pTimeStart, pTimeStop, pivotTimeStart, pivotTimeStop, updateTimeStart, updateTimeStop;
hipEvent_t trim1TimeStart, trim1TimeStop, bfsTimeStart, bfsTimeStop;
hipEventCreate(&bTimeStart);
hipEventCreate(&bTimeStop);

hipEventCreate(&pTimeStart);
hipEventCreate(&pTimeStop);

hipEventCreate(&pivotTimeStart);
hipEventCreate(&pivotTimeStop);

hipEventCreate(&trim1TimeStart);
hipEventCreate(&trim1TimeStop);

hipEventCreate(&updateTimeStart);
hipEventCreate(&updateTimeStop);

hipEventCreate(&bfsTimeStart);
hipEventCreate(&bfsTimeStop);
#endif


#ifdef _DEBUG
hipEventRecord(trim1TimeStart, 0);
#endif

//-----------Trimming-------------------------------------->
        if(t1)
            trim1<<<grid, threads>>>( d_range, d_tags, d_Fc, d_Fr, d_Bc, d_Br, RSize, d_terminatef);

#ifdef _DEBUG
hipEventRecord(trim1TimeStop, 0);
hipEventSynchronize(trim1TimeStop);
hipEventElapsedTime(&temp, trim1TimeStart, trim1TimeStop);
trim1Time+=temp;
#endif

//-----------Choose pivots--------------------------------->
#ifdef _DEBUG
hipEventRecord(pivotTimeStart, 0);
#endif

        CUDA_SAFE_CALL( hipMemset( d_pivots, 0, sizeof(uint32_t) ));
        pollForFirstPivot<<<grid, threads>>>( d_tags, RSize, d_pivots, d_Fr, d_Br);
        selectFirstPivot<<<grid, threads>>>( d_tags, RSize, d_pivots);

#ifdef _DEBUG
hipEventRecord(pivotTimeStop, 0);
hipEventSynchronize(pivotTimeStop);

hipEventElapsedTime(&temp, pivotTimeStart, pivotTimeStop);
pivotTime+=temp;
#endif


#ifdef _DEBUG
hipEventRecord(bfsTimeStart, 0);
#endif

        do{//Forward and Backward reachability
            FWD_iterations++;
            BWD_iterations++;

            CUDA_SAFE_CALL( hipMemset( (void *)d_terminatef, true, sizeof(bool) ));
            CUDA_SAFE_CALL( hipMemset((void *)d_terminateb, true, sizeof(bool) ));

            switch(warpSize){
                case 1:
                    fwd_warp<1><<<gridfb, threads>>>( d_Fc, d_Fr, d_range, d_tags, RSize, d_terminatef);
                    bwd_warp<1><<<gridfb, threads>>>( d_Bc, d_Br, d_range, d_tags, RSize, d_terminateb);
                    break;

                case 2:
                    fwd_warp<2><<<gridfb, threads>>>( d_Fc, d_Fr, d_range, d_tags, RSize, d_terminatef);
                    bwd_warp<2><<<gridfb, threads>>>( d_Bc, d_Br, d_range, d_tags, RSize, d_terminateb);
                    break;

                case 4:
                    fwd_warp<4><<<gridfb, threads>>>( d_Fc, d_Fr, d_range, d_tags, RSize, d_terminatef);
                    bwd_warp<4><<<gridfb, threads>>>( d_Bc, d_Br, d_range, d_tags, RSize, d_terminateb);
                    break;

                case 8:
                    fwd_warp<8><<<gridfb, threads>>>( d_Fc, d_Fr, d_range, d_tags, RSize, d_terminatef);
                    bwd_warp<8><<<gridfb, threads>>>( d_Bc, d_Br, d_range, d_tags, RSize, d_terminateb);
                    break;

                case 16:
                    fwd_warp<16><<<gridfb, threads>>>( d_Fc, d_Fr, d_range, d_tags, RSize, d_terminatef);
                    bwd_warp<16><<<gridfb, threads>>>( d_Bc, d_Br, d_range, d_tags, RSize, d_terminateb);
                    break;

                case 32:
                    fwd_warp<32><<<gridfb, threads>>>( d_Fc, d_Fr, d_range, d_tags, RSize, d_terminatef);
                    bwd_warp<32><<<gridfb, threads>>>( d_Bc, d_Br, d_range, d_tags, RSize, d_terminateb);
                    break;

            }

            CUDA_SAFE_CALL( hipMemcpy( &terminatef, (const void *)d_terminatef, sizeof(bool), hipMemcpyDeviceToHost ));
            CUDA_SAFE_CALL( hipMemcpy( &terminateb, (const void *)d_terminateb, sizeof(bool), hipMemcpyDeviceToHost ));
        }while(!terminatef && !terminateb);

        while(!terminatef){//Forward reachability
            FWD_iterations++;

            CUDA_SAFE_CALL( hipMemset( (void *)d_terminatef, true, sizeof(bool) ));
            switch(warpSize){
                case 1:
                    fwd_warp<1><<<gridfb, threads>>>( d_Fc, d_Fr, d_range, d_tags, RSize, d_terminatef);
                    break;

                case 2:
                    fwd_warp<2><<<gridfb, threads>>>( d_Fc, d_Fr, d_range, d_tags, RSize, d_terminatef);
                    break;

                case 4:
                    fwd_warp<4><<<gridfb, threads>>>( d_Fc, d_Fr, d_range, d_tags, RSize, d_terminatef);
                    break;

                case 8:
                    fwd_warp<8><<<gridfb, threads>>>( d_Fc, d_Fr, d_range, d_tags, RSize, d_terminatef);
                    break;

                case 16:
                    fwd_warp<16><<<gridfb, threads>>>( d_Fc, d_Fr, d_range, d_tags, RSize, d_terminatef);
                    break;

                case 32:
                    fwd_warp<32><<<gridfb, threads>>>( d_Fc, d_Fr, d_range, d_tags, RSize, d_terminatef);
                    break;
            }

            CUDA_SAFE_CALL( hipMemcpy( &terminatef, (const void *)d_terminatef, sizeof(bool), hipMemcpyDeviceToHost ));
        }

         while(!terminateb){//Backward reachability
            BWD_iterations++;

            CUDA_SAFE_CALL( hipMemset((void *)d_terminateb, true, sizeof(bool) ));

            switch(warpSize){
                case 1:
                    bwd_warp<1><<<gridfb, threads>>>( d_Bc, d_Br, d_range, d_tags, RSize, d_terminateb);
                    break;

                case 2:
                    bwd_warp<2><<<gridfb, threads>>>( d_Bc, d_Br, d_range, d_tags, RSize, d_terminateb);
                    break;

                case 4:
                    bwd_warp<4><<<gridfb, threads>>>( d_Bc, d_Br, d_range, d_tags, RSize, d_terminateb);
                    break;

                case 8:
                    bwd_warp<8><<<gridfb, threads>>>( d_Bc, d_Br, d_range, d_tags, RSize, d_terminateb);
                    break;

                case 16:
                    bwd_warp<16><<<gridfb, threads>>>( d_Bc, d_Br, d_range, d_tags, RSize, d_terminateb);
                    break;

                case 32:
                    bwd_warp<32><<<gridfb, threads>>>( d_Bc, d_Br, d_range, d_tags, RSize, d_terminateb);
                    break;
            }

            CUDA_SAFE_CALL( hipMemcpy( &terminateb, (const void *)d_terminateb, sizeof(bool), hipMemcpyDeviceToHost ));
        }

#ifdef _DEBUG
hipEventRecord(bfsTimeStop, 0);
hipEventSynchronize(bfsTimeStop);

hipEventElapsedTime(&temp, bfsTimeStart, bfsTimeStop);
bfsTime+=temp;
#endif

#ifdef _DEBUG
hipEventRecord(updateTimeStart, 0);
#endif

        update<<<grid, threads>>>(d_range, d_tags, RSize, d_terminatef);

#ifdef _DEBUG
hipEventRecord(updateTimeStop, 0);
hipEventSynchronize(updateTimeStop);

hipEventElapsedTime(&temp, updateTimeStart, updateTimeStop);
updateTime+=temp;
#endif

//-----------Main algorithm-------------------------------->
    while ( true ) {
        iterations++;
        //cout<<"\nIteration : "<<iterations<<endl;
#ifdef _DEBUG
hipEventRecord(pTimeStart, 0);
#endif

        assignUniqueRange<<<grid, threads>>>(d_range, d_tags, RSize);

        do{
            CUDA_SAFE_CALL( hipMemset((void *)d_terminatef, true, sizeof(bool) ));
            colorPropagation<<<grid, threads>>>( d_Fc, d_Fr, d_range, d_tags, RSize, d_terminatef);
            CUDA_SAFE_CALL( hipMemcpy( &terminatef, (const void *)d_terminatef, sizeof(bool), hipMemcpyDeviceToHost ));
        }while(!terminatef);

#ifdef _DEBUG
hipEventRecord(pTimeStop, 0);
hipEventSynchronize(pTimeStop);

hipEventElapsedTime(&temp, pTimeStart, pTimeStop);
pTime+=temp;
#endif

#ifdef _DEBUG
hipEventRecord(bTimeStart, 0);
#endif
        selectPivotColoring<<<grid, threads>>>(d_range, d_tags, RSize);

        do{//Forward reachability
            CUDA_SAFE_CALL( hipMemset( (void *)d_terminatef, true, sizeof(bool) ));
            fwdColoring<<<grid, threads>>>( d_Fc, d_Fr, d_range, d_tags, RSize, d_terminatef);
            CUDA_SAFE_CALL( hipMemcpy( &terminatef, (const void *)d_terminatef, sizeof(bool), hipMemcpyDeviceToHost ));
        }while(!terminatef);

        CUDA_SAFE_CALL( hipMemset( (void *)d_terminatef, true, sizeof(bool) ));
        updateColoring<<<grid, threads>>>(d_tags, RSize, d_terminatef);
        CUDA_SAFE_CALL( hipMemcpy( &terminatef, (const void *)d_terminatef, sizeof(bool), hipMemcpyDeviceToHost ));
        if (terminatef)
            break; //only way out

#ifdef _DEBUG
hipEventRecord(bTimeStop, 0);
hipEventSynchronize(bTimeStop);

hipEventElapsedTime(&temp, bTimeStart, bTimeStop);
bTime+=temp;
#endif

    }
//<----------Main algorithm---------------------------------

    //SCC extraction
    CUDA_SAFE_CALL( hipMemcpy(tags, d_tags, sizeof(uint8_t) * (RSize + 1), hipMemcpyDeviceToHost ));
    uint32_t numberOf1Sccs = 0;
    uint32_t numberOfPivotSccs = 0;
    uint32_t numberOfSccs = 0;

    for(uint32_t i=1;i<=RSize;i++)
        if(isTrim1(tags[i]))
            numberOf1Sccs++;
        else if(isPivot(tags[i]))
            numberOfPivotSccs++;

    numberOfSccs = numberOf1Sccs + numberOfPivotSccs;

    hipEventRecord(sccTimeStop, 0);
    hipEventSynchronize(sccTimeStop);
    hipEventElapsedTime(&sccTime, sccTimeStart, sccTimeStop);

    //printf(", %u, %d, %d", iterations, FWD_iterations , BWD_iterations);

#ifdef _DEBUG
printf(", %f", bfsTime);
printf(", %f", pTime);
printf(", %f", bTime);
printf(", %f", trim1Time);
printf(", %f", pivotTime);
printf(", %f", updateTime);
#endif

    printf("\nNumber Of Sccs : %d", numberOfSccs);
    printf("\nTime : %f", sccTime );

    CUDA_SAFE_CALL( hipFree( d_Fc ));
    CUDA_SAFE_CALL( hipFree( d_Fr ));
    CUDA_SAFE_CALL( hipFree( d_Bc ));
    CUDA_SAFE_CALL( hipFree( d_Br ));
    CUDA_SAFE_CALL( hipFree( d_range));
    CUDA_SAFE_CALL( hipFree( d_tags));
    CUDA_SAFE_CALL( hipFree( d_pivots ));
    CUDA_SAFE_CALL( hipFree( (void *)d_terminatef));
    CUDA_SAFE_CALL( hipFree( (void *)d_terminateb));

    hipEventDestroy(sccTimeStart);
    hipEventDestroy(sccTimeStop);

#ifdef _DEBUG
hipEventDestroy(bTimeStart);
hipEventDestroy(bTimeStop);
hipEventDestroy(pTimeStart);
hipEventDestroy(pTimeStop);
hipEventDestroy(trim1TimeStart);
hipEventDestroy(trim1TimeStop);
hipEventDestroy(bfsTimeStart);
hipEventDestroy(bfsTimeStop);
hipEventDestroy(pivotTimeStart);
hipEventDestroy(pivotTimeStop);
hipEventDestroy(updateTimeStart);
hipEventDestroy(updateTimeStop);
#endif

    return;
}


void vSlota(uint32_t CSize, uint32_t RSize, uint32_t *Fc, uint32_t *Fr, uint32_t * Bc, uint32_t * Br, bool t1, bool t2){
    //Set the device which exclusively used by this program
    hipSetDevice(7);

    float sccTime=0;
    hipEvent_t sccTimeStart, sccTimeStop;
    hipEventCreate(&sccTimeStart);
    hipEventCreate(&sccTimeStop);
    hipEventRecord(sccTimeStart, 0);

//-----------GPU initialization---------------------------->
	uint32_t* d_Fr = NULL;
    uint32_t* d_Br = NULL;
	uint32_t* d_Fc = NULL;
    uint32_t* d_Bc = NULL;
    uint32_t* d_pivots = NULL;

	uint32_t* d_range = NULL;
    uint8_t* d_tags = NULL;
    uint8_t* tags = new uint8_t[RSize+1];

    bool volatile* d_terminatef = NULL;
    bool terminatef = false;

    bool volatile* d_terminateb = NULL;
    bool terminateb = false;

	int FWD_iterations = 0;
    int BWD_iterations = 0;
	uint32_t iterations = 0;

    const uint32_t max_pivot_count = 1;

	hipError_t e1, e2, e3, e4, e5, e6, e7, e8, e9;
	CUDA_SAFE_CALL( e1 = hipMalloc( (void**) &d_Fc, CSize * sizeof(uint32_t) ));
	CUDA_SAFE_CALL( e2 = hipMalloc( (void**) &d_Fr, (RSize + 2) * sizeof(uint32_t) ));
	CUDA_SAFE_CALL( e3 = hipMalloc( (void**) &d_Bc, CSize * sizeof(uint32_t) ));
	CUDA_SAFE_CALL( e4 = hipMalloc( (void**) &d_Br, (RSize + 2) * sizeof(uint32_t) ));
	CUDA_SAFE_CALL( e5 = hipMalloc( (void**) &d_range,  (RSize + 1) * sizeof(uint32_t)));
    CUDA_SAFE_CALL( e6 = hipMalloc( (void**) &d_tags,  (RSize + 1) * sizeof(uint8_t)));
    CUDA_SAFE_CALL( e7 = hipMalloc( (void**) &d_pivots, max_pivot_count * sizeof(uint32_t) ));
    CUDA_SAFE_CALL( e8 = hipMalloc( (void**) &d_terminatef, sizeof(bool) ));
    CUDA_SAFE_CALL( e9 = hipMalloc( (void**) &d_terminateb, sizeof(bool) ));

	if (e1 == hipErrorOutOfMemory || e2 == hipErrorOutOfMemory ||
		e3 == hipErrorOutOfMemory || e4 == hipErrorOutOfMemory ||
		e5 == hipErrorOutOfMemory || e6 == hipErrorOutOfMemory ||
        e7 == hipErrorOutOfMemory || e8 == hipErrorOutOfMemory || e9 == hipErrorOutOfMemory) {
		throw "Error: Not enough memory on GPU\n";
	}

	CUDA_SAFE_CALL( hipMemcpy( d_Fc, Fc, CSize * sizeof(uint32_t), hipMemcpyHostToDevice ));
	CUDA_SAFE_CALL( hipMemcpy( d_Fr, Fr, (RSize + 2) * sizeof(uint32_t), hipMemcpyHostToDevice ));
	CUDA_SAFE_CALL( hipMemcpy( d_Bc, Bc, CSize * sizeof(uint32_t), hipMemcpyHostToDevice ));
	CUDA_SAFE_CALL( hipMemcpy( d_Br, Br, (RSize + 2) * sizeof(uint32_t), hipMemcpyHostToDevice ));
	
    CUDA_SAFE_CALL( hipMemset( d_range, 0, (RSize + 1) * sizeof(uint32_t)));
    CUDA_SAFE_CALL( hipMemset( d_tags, 0, (RSize + 1) * sizeof(uint8_t)));

    //for vertex-to-thread mapping
    dim3 grid;
    if((RSize + BLOCKSIZE - 1)/BLOCKSIZE > MaxXDimOfGrid) {
        int dim = ceill(sqrt(RSize / BLOCKSIZE));
        grid.x = dim;
        grid.y = dim;
        grid.z = 1;
    }else{
        grid.x = (RSize + BLOCKSIZE - 1)/BLOCKSIZE;
        grid.y = 1;
        grid.z = 1;
    }


	dim3 threads(BLOCKSIZE, 1, 1);

#ifdef _DEBUG
float pivotTime = 0, temp = 0, pTime = 0, bTime = 0, trim1Time = 0, updateTime = 0, bfsTime = 0;
hipEvent_t pTimeStart, pTimeStop, bTimeStart, bTimeStop, pivotTimeStart, pivotTimeStop, updateTimeStart, updateTimeStop;
hipEvent_t trim1TimeStart, trim1TimeStop, bfsTimeStart, bfsTimeStop;
hipEventCreate(&pTimeStart);
hipEventCreate(&pTimeStop);

hipEventCreate(&bTimeStart);
hipEventCreate(&bTimeStop);

hipEventCreate(&pivotTimeStart);
hipEventCreate(&pivotTimeStop);

hipEventCreate(&trim1TimeStart);
hipEventCreate(&trim1TimeStop);

hipEventCreate(&updateTimeStart);
hipEventCreate(&updateTimeStop);

hipEventCreate(&bfsTimeStart);
hipEventCreate(&bfsTimeStop);
#endif


#ifdef _DEBUG
hipEventRecord(trim1TimeStart, 0);
#endif

//-----------Trimming-------------------------------------->
        if(t1)
            trim1<<<grid, threads>>>( d_range, d_tags, d_Fc, d_Fr, d_Bc, d_Br, RSize, d_terminatef);

#ifdef _DEBUG
hipEventRecord(trim1TimeStop, 0);
hipEventSynchronize(trim1TimeStop);
hipEventElapsedTime(&temp, trim1TimeStart, trim1TimeStop);
trim1Time+=temp;
#endif

//-----------Choose pivots--------------------------------->
#ifdef _DEBUG
hipEventRecord(pivotTimeStart, 0);
#endif

        CUDA_SAFE_CALL( hipMemset( d_pivots, 0, sizeof(uint32_t) ));
        pollForFirstPivot<<<grid, threads>>>( d_tags, RSize, d_pivots, d_Fr, d_Br);
        selectFirstPivot<<<grid, threads>>>( d_tags, RSize, d_pivots);

#ifdef _DEBUG
hipEventRecord(pivotTimeStop, 0);
hipEventSynchronize(pivotTimeStop);

hipEventElapsedTime(&temp, pivotTimeStart, pivotTimeStop);
pivotTime+=temp;
#endif


#ifdef _DEBUG
hipEventRecord(bfsTimeStart, 0);
#endif

        do{//Forward and Backward reachability
            FWD_iterations++;
            BWD_iterations++;

            CUDA_SAFE_CALL( hipMemset( (void *)d_terminatef, true, sizeof(bool) ));
            CUDA_SAFE_CALL( hipMemset((void *)d_terminateb, true, sizeof(bool) ));
            fwd<<<grid, threads>>>( d_Fc, d_Fr, d_range, d_tags, RSize, d_terminatef);
            bwd<<<grid, threads>>>( d_Bc, d_Br, d_range, d_tags, RSize, d_terminateb);
            CUDA_SAFE_CALL( hipMemcpy( &terminatef, (const void *)d_terminatef, sizeof(bool), hipMemcpyDeviceToHost ));
            CUDA_SAFE_CALL( hipMemcpy( &terminateb, (const void *)d_terminateb, sizeof(bool), hipMemcpyDeviceToHost ));
        }while(!terminatef && !terminateb);

        while(!terminatef){//Forward reachability
            FWD_iterations++;

            CUDA_SAFE_CALL( hipMemset( (void *)d_terminatef, true, sizeof(bool) ));
            fwd<<<grid, threads>>>( d_Fc, d_Fr, d_range, d_tags, RSize, d_terminatef);
            CUDA_SAFE_CALL( hipMemcpy( &terminatef, (const void *)d_terminatef, sizeof(bool), hipMemcpyDeviceToHost ));
        }

         while(!terminateb){//Backward reachability
            BWD_iterations++;

            CUDA_SAFE_CALL( hipMemset((void *)d_terminateb, true, sizeof(bool) ));
            bwd<<<grid, threads>>>( d_Bc, d_Br, d_range, d_tags, RSize, d_terminateb);
            CUDA_SAFE_CALL( hipMemcpy( &terminateb, (const void *)d_terminateb, sizeof(bool), hipMemcpyDeviceToHost ));
        }

#ifdef _DEBUG
hipEventRecord(bfsTimeStop, 0);
hipEventSynchronize(bfsTimeStop);

hipEventElapsedTime(&temp, bfsTimeStart, bfsTimeStop);
bfsTime+=temp;
#endif

#ifdef _DEBUG
hipEventRecord(updateTimeStart, 0);
#endif

        update<<<grid, threads>>>(d_range, d_tags, RSize, d_terminatef);

#ifdef _DEBUG
hipEventRecord(updateTimeStop, 0);
hipEventSynchronize(updateTimeStop);

hipEventElapsedTime(&temp, updateTimeStart, updateTimeStop);
updateTime+=temp;
#endif

//-----------Main algorithm-------------------------------->
	while ( true ) {
		iterations++;
        //cout<<"\nIteration : "<<iterations<<endl;
#ifdef _DEBUG
hipEventRecord(pTimeStart, 0);
#endif
        assignUniqueRange<<<grid, threads>>>(d_range, d_tags, RSize);

        do{
            CUDA_SAFE_CALL( hipMemset((void *)d_terminatef, true, sizeof(bool) ));
            colorPropagation<<<grid, threads>>>( d_Fc, d_Fr, d_range, d_tags, RSize, d_terminatef);
            CUDA_SAFE_CALL( hipMemcpy( &terminatef, (const void *)d_terminatef, sizeof(bool), hipMemcpyDeviceToHost ));
        }while(!terminatef);

#ifdef _DEBUG
hipEventRecord(pTimeStop, 0);
hipEventSynchronize(pTimeStop);

hipEventElapsedTime(&temp, pTimeStart, pTimeStop);
pTime+=temp;
#endif

#ifdef _DEBUG
hipEventRecord(bTimeStart, 0);
#endif

        selectPivotColoring<<<grid, threads>>>(d_range, d_tags, RSize);

        do{//Forward reachability
            CUDA_SAFE_CALL( hipMemset( (void *)d_terminatef, true, sizeof(bool) ));
            fwdColoring<<<grid, threads>>>( d_Fc, d_Fr, d_range, d_tags, RSize, d_terminatef);
            CUDA_SAFE_CALL( hipMemcpy( &terminatef, (const void *)d_terminatef, sizeof(bool), hipMemcpyDeviceToHost ));
        }while(!terminatef);

        CUDA_SAFE_CALL( hipMemset( (void *)d_terminatef, true, sizeof(bool) ));
        updateColoring<<<grid, threads>>>(d_tags, RSize, d_terminatef);
        CUDA_SAFE_CALL( hipMemcpy( &terminatef, (const void *)d_terminatef, sizeof(bool), hipMemcpyDeviceToHost ));
        if (terminatef)
            break; //only way out

#ifdef _DEBUG
hipEventRecord(bTimeStop, 0);
hipEventSynchronize(bTimeStop);

hipEventElapsedTime(&temp, bTimeStart, bTimeStop);
bTime+=temp;
#endif
	}
//<----------Main algorithm---------------------------------

    //SCC extraction
    CUDA_SAFE_CALL( hipMemcpy(tags, d_tags, sizeof(uint8_t) * (RSize + 1), hipMemcpyDeviceToHost ));
    uint32_t numberOf1Sccs = 0;
    uint32_t numberOfPivotSccs = 0;
    uint32_t numberOfSccs = 0;

    for(uint32_t i=1;i<=RSize;i++)
        if(isTrim1(tags[i]))
            numberOf1Sccs++;
        else if(isPivot(tags[i]))
            numberOfPivotSccs++;

    numberOfSccs = numberOf1Sccs + numberOfPivotSccs;

	hipEventRecord(sccTimeStop, 0);
    hipEventSynchronize(sccTimeStop);
    hipEventElapsedTime(&sccTime, sccTimeStart, sccTimeStop);

    //printf(", %u, %d, %d", iterations, FWD_iterations , BWD_iterations);

#ifdef _DEBUG
printf(", %f", bfsTime);
printf(", %f", pTime);
printf(", %f", bTime);
printf(", %f", trim1Time);
printf(", %f", pivotTime);
printf(", %f", updateTime);
#endif

    printf("\nNumber Of Sccs : %d", numberOfSccs);
    printf("\nTime : %f", sccTime );

	CUDA_SAFE_CALL( hipFree( d_Fc ));
	CUDA_SAFE_CALL( hipFree( d_Fr ));
	CUDA_SAFE_CALL( hipFree( d_Bc ));
	CUDA_SAFE_CALL( hipFree( d_Br ));
	CUDA_SAFE_CALL( hipFree( d_range));
    CUDA_SAFE_CALL( hipFree( d_tags));
	CUDA_SAFE_CALL( hipFree( d_pivots ));
	CUDA_SAFE_CALL( hipFree( (void *)d_terminatef));
    CUDA_SAFE_CALL( hipFree( (void *)d_terminateb));

	hipEventDestroy(sccTimeStart);
    hipEventDestroy(sccTimeStop);

#ifdef _DEBUG
hipEventDestroy(bTimeStart);
hipEventDestroy(bTimeStop);
hipEventDestroy(pTimeStart);
hipEventDestroy(pTimeStop);
hipEventDestroy(trim1TimeStart);
hipEventDestroy(trim1TimeStop);
hipEventDestroy(pivotTimeStart);
hipEventDestroy(pivotTimeStop);
hipEventDestroy(updateTimeStart);
hipEventDestroy(updateTimeStop);
hipEventDestroy(bfsTimeStart);
hipEventDestroy(bfsTimeStop);
#endif

	return;
}
