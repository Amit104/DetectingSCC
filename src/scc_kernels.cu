#include "hip/hip_runtime.h"
#include "scc_kernels.h"


__global__ void selectPivots(const uint32_t *range, uint8_t *tags, const uint32_t num_rows, const uint32_t *pivot_field, const int max_pivot_count){

    uint32_t row = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x + 1;
    uint8_t myTag;

    if (row > num_rows || isRangeSet(myTag = tags[row]))
        return;

    if( pivot_field[ range[row] % max_pivot_count] == row ) {
        myTag = 0;
        setForwardVisitedBit(&myTag);
        setBackwardVisitedBit(&myTag);
        setPivot(&myTag);
        tags[row] = myTag;
    }
}

__global__ void selectPivotsLocal(const uint32_t *range, uint8_t *tags, const uint32_t num_rows, uint32_t** pivot_field, const int max_pivot_count, uint32_t *Pr, bool *auxRange){

    uint32_t row = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x + 1;
    uint8_t myTag;

    if (row > num_rows || isRangeSet(myTag = tags[row]) || auxRange[row])
        return;
    uint32_t myPr = Pr[row];
    if( pivot_field[myPr][range[row] % max_pivot_count] == row ) {
        myTag = 0;
        //printf("Selected pivot: %d\n", row);
        setForwardVisitedBit(&myTag);
        setBackwardVisitedBit(&myTag);
        setPivot(&myTag);
        tags[row] = myTag;
    }
}

__global__ void selectPivotsLocalNew(const uint32_t *range, uint8_t *tags, const uint32_t num_rows, uint32_t** pivot_field, const int max_pivot_count, uint32_t *Pr, bool *auxRange){

    uint32_t row = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x + 1;
    uint8_t myTag;

    if (row > num_rows || isRangeSet(myTag = tags[row]) || auxRange[row])
        return;
    uint32_t myPr = Pr[row];
    if( pivot_field[myPr][range[row]] == row ) {
        myTag = 0;
        //printf("Selected pivot: %d\n", row);
        setForwardVisitedBit(&myTag);
        setBackwardVisitedBit(&myTag);
        setPivot(&myTag);
        tags[row] = myTag;
    }
}

__global__ void pollForPivots(const uint32_t *range, const uint8_t *tags, const uint32_t num_rows, uint32_t* pivot_field, const int max_pivot_count, const uint32_t *Fr, const uint32_t *Br){

    uint32_t row = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x + 1;

    if (row > num_rows || isRangeSet(tags[row]))
        return;

    uint32_t index = range[row];

    uint32_t oldRow = pivot_field[index % max_pivot_count];
    uint32_t oldDegree = (Fr[oldRow+1] - Fr[oldRow]) * (Br[oldRow+1] - Br[oldRow]);
    uint32_t newDegree = (Fr[row+1] - Fr[row]) * (Br[row+1] - Br[row]);

    if(newDegree > oldDegree)
    {
        pivot_field[ index % max_pivot_count ] = row;
    }
}


__global__ void pollForPivotsLocalNew(const uint32_t *range, const uint8_t *tags, const uint32_t num_rows, uint32_t** pivot_field, const int max_pivot_count, const uint32_t *Fr, const uint32_t *Br, uint32_t *Pr, bool volatile *terminate, bool *auxRange){

    uint32_t row = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x + 1;

    if (row > num_rows || isRangeSet(tags[row]) || auxRange[row])
        return;

    uint32_t index = range[row];
    uint32_t myPr = Pr[row];
    //printf("$%d %d\n", myPr, index);
    uint32_t oldRow = pivot_field[myPr][index];

    uint32_t oldDegree = (Fr[oldRow+1] - Fr[oldRow]) * (Br[oldRow+1] - Br[oldRow]);
    uint32_t newDegree = (Fr[row+1] - Fr[row]) * (Br[row+1] - Br[row]);
    //printf("%d %d %d %d %d %d\n", row, myPr, index, index % max_pivot_count, newDegree, oldRow);
    if(newDegree > oldDegree)
    {
        *terminate = false;
        pivot_field[myPr][index] = row;
        //printf("Updated %d\n", row);
        //printf("Update: %d, %d, %d, %d\n", myPr, index % max_pivot_count, row, pivot_field[myPr][index % max_pivot_count]);
    }
}

__global__ void pollForPivotsLocal(const uint32_t *range, const uint8_t *tags, const uint32_t num_rows, uint32_t** pivot_field, const int max_pivot_count, const uint32_t *Fr, const uint32_t *Br, uint32_t *Pr, bool volatile *terminate, bool *auxRange){

    uint32_t row = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x + 1;

    if (row > num_rows || isRangeSet(tags[row]) || auxRange[row])
        return;

    uint32_t index = range[row];
    uint32_t myPr = Pr[row];
    uint32_t oldRow = pivot_field[myPr][index % max_pivot_count];

    uint32_t oldDegree = (Fr[oldRow+1] - Fr[oldRow]) * (Br[oldRow+1] - Br[oldRow]);
    uint32_t newDegree = (Fr[row+1] - Fr[row]) * (Br[row+1] - Br[row]);
    //printf("%d %d %d %d %d %d\n", row, myPr, index, index % max_pivot_count, newDegree, oldRow);
    if(newDegree > oldDegree)
    {
        *terminate = false;
        pivot_field[myPr][index % max_pivot_count] = row;
        //printf("Updated %d\n", row);
        //printf("Update: %d, %d, %d, %d\n", myPr, index % max_pivot_count, row, pivot_field[myPr][index % max_pivot_count]);
    }
}

__global__ void update(uint32_t *range, uint8_t *tags, const uint32_t num_rows, bool volatile *terminate){

    uint32_t row = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x + 1;
    uint8_t myTag;

    if (row > num_rows || isRangeSet(myTag = tags[row]))
        return;

    if ( isForwardVisited(myTag) && isBackwardVisited(myTag)){
        rangeSet(&tags[row]);
        //printf("Range set %d\n", row);
    }
    else{
        *terminate = false;
        uint32_t index = 3 * range[row] + (uint32_t)isForwardVisited(myTag) + ((uint32_t)isBackwardVisited(myTag) << 1);
        range[row] = index;
        tags[row] = 0;
    }
}

__global__ void updateLocal(uint32_t *range, uint8_t *tags, const uint32_t num_rows, bool volatile *terminate, bool *auxRange){

    uint32_t row = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x + 1;
    uint8_t myTag;

    if (row > num_rows || isRangeSet(myTag = tags[row]) || auxRange[row])
        return;

    if ( isForwardVisited(myTag) && isBackwardVisited(myTag)){
        //rangeSet(&tags[row]);
        auxRange[row] = true;
        //printf("AuxRange set %d\n", row);
    }
    else{
        *terminate = false;
        uint32_t index = 3 * range[row] + (uint32_t)isForwardVisited(myTag) + ((uint32_t)isBackwardVisited(myTag) << 1);
        range[row] = index;
        tags[row] = 0;
    }
}

__global__ void resetTag(uint32_t *range, uint8_t *tags, const uint32_t num_rows, const int i){

    uint32_t row = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x + 1;
    uint8_t myTag;

    if (row > num_rows || isRangeSet(myTag = tags[row]))
        return;
    if(row == i)
    {
        myTag = 0;
        setForwardVisitedBit(&myTag);
        setBackwardVisitedBit(&myTag);
        tags[row] = myTag;
        //printf("Tagging 1 for %d\n", row);
    }
    else
    {
        tags[row] = 0;
        //printf("Tagging 0 for %d\n", row);
    }
}

__global__ void trim1(const uint32_t *range, uint8_t *tags, const uint32_t *Fc, const uint32_t *Fr, const uint32_t *Bc, const uint32_t *Br, const uint32_t num_rows, bool volatile *terminate){

	uint32_t row = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x + 1;
	uint8_t myTag;

    if (row > num_rows || isRangeSet(myTag = tags[row]))
        return;

    uint32_t myRange = range[row];

	uint32_t cnt = Br[row + 1] - Br[row];
    const uint32_t *nbrs = &Bc[Br[row]];

	bool eliminate = true;
	for(uint32_t i = 0; i < cnt; i++){
	    uint32_t index = nbrs[i];

		if ( !isRangeSet(tags[index]) && range[index] == myRange){
			eliminate = false;
            break;
        }
	}

	if ( !eliminate ) {
		eliminate = true;
		cnt = Fr[row + 1] - Fr[row];
        nbrs = &Fc[Fr[row]];

		for(uint32_t i = 0; i < cnt; i++){
	        uint32_t index = nbrs[i];

			if ( !isRangeSet(tags[index]) && range[index] == myRange){
				eliminate = false;
                break;
            }
		}
	}

	if ( eliminate ) {
		rangeSet(&myTag);
        //printf("Range is set for %d\n", row);
        setTrim1(&myTag);
        tags[row] = myTag;
		*terminate = false;
	}
	return;
}


__global__ void trim2(const uint32_t *range, uint8_t *tags, const uint32_t *Fc, const uint32_t *Fr, const uint32_t *Bc, const uint32_t *Br, const uint32_t num_rows){

    uint32_t row = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x + 1;

    if (row > num_rows || isRangeSet(tags[row]))
        return;

    uint32_t myRange = range[row];
    uint32_t cnt = Br[row + 1] - Br[row];
    const uint32_t *nbrs = &Bc[Br[row]];
    uint32_t inDegree = 0;
    uint32_t k = 0;  //other neighbour

    bool eliminate = false;
    for(uint32_t i = 0; i < cnt; i++){
        uint32_t index = nbrs[i];

        if (!isRangeSet(tags[index]) && range[index] == myRange){
            inDegree++;

            if(inDegree == 2)
                break;

            k = index;
        }
    }

    if(inDegree == 1){
        cnt = Fr[row + 1] - Fr[row];
        nbrs = &Fc[Fr[row]];

        for(uint32_t i = 0; i < cnt; i++){
            uint32_t index = nbrs[i];

            if(index == k){

                uint32_t kCnt = Br[k + 1] - Br[k];
                const uint32_t *kNbrs = &Bc[Br[k]];
                uint32_t kRange = range[k];
                inDegree = 0;

                for(uint32_t j = 0; j < kCnt; j++){
                    uint32_t tindex = kNbrs[j];

                    if(!isRangeSet(tags[tindex]) && range[tindex] == kRange){
                        inDegree++;

                        if(inDegree==2)
                            break;
                    }
                }

                if(inDegree == 1)
                    eliminate = true;

                break;
            }
        }
    }


    if(!eliminate){
        cnt = Fr[row + 1] - Fr[row];
        nbrs = &Fc[Fr[row]];
        inDegree=0;
        k = 0;

        for( uint32_t i = 0; i < cnt; i++ ){
            uint32_t index = nbrs[i];

            if ( !isRangeSet(tags[index]) && range[index] == myRange){
                inDegree++;

                if(inDegree == 2)
                    break;

                k = index;
            }
        }

        if(inDegree == 1){
            cnt = Br[row + 1] - Br[row];
            nbrs = &Bc[Br[row]];

            for(uint32_t i = 0; i < cnt; i++){
                uint32_t index = nbrs[i];

                if(index == k){

                    uint32_t kCnt = Fr[k + 1] - Fr[k];
                    const uint32_t *kNbrs = &Fc[Fr[k]];
                    uint32_t kRange = range[k];
                    inDegree = 0;

                    for(uint32_t j = 0; j < kCnt; j++){
                        uint32_t tindex = kNbrs[j];

                        if(!isRangeSet(tags[tindex]) && range[tindex] == kRange){
                            inDegree++;

                            if(inDegree==2)
                                break;
                        }
                    }

                    if(inDegree == 1)
                        eliminate = true;

                    break;
                }
            }
        }
    }

    if(eliminate){
        uint32_t temp = min(row, k);
        rangeSet(&tags[row]);
        rangeSet(&tags[k]);
        setTrim2(&tags[temp]); //Only one of the two will be set as pivot for 2-SCC
    }
    return;
}


__global__ void fwd(const uint32_t *Fc, const uint32_t *Fr, const uint32_t *range, uint8_t *tags, const uint32_t num_rows, bool volatile *terminate){

	uint32_t row = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x + 1;
	uint8_t myTag;

    if (row > num_rows || isRangeSet(myTag = tags[row]) || isForwardPropagate(myTag) || !isForwardVisited(myTag))
        return;

    uint32_t myRange = range[row];
	uint32_t cnt = Fr[row + 1] - Fr[row];
    const uint32_t *nbrs = &Fc[Fr[row]];

	bool end = true;
	for ( uint32_t i = 0; i < cnt; i++ ) {
        uint32_t index = nbrs[i];
        uint8_t nbrTag = tags[index];

		if(isRangeSet(nbrTag) || isForwardVisited(nbrTag) || range[index] != myRange)
			continue;

		setForwardVisitedBit(&tags[index]);
		end = false;
	}
	setForwardPropagateBit(&tags[row]);
	if (!end)
		*terminate = false;
}

__global__ void fwdLocal(const uint32_t *Fc, const uint32_t *Fr, const uint32_t *range, uint8_t *tags, uint32_t *Pr, const uint32_t num_rows, bool volatile *terminate, bool *auxRange){

	uint32_t row = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x + 1;
	uint8_t myTag;

    if (row > num_rows || isRangeSet(myTag = tags[row]) || isForwardPropagate(myTag) || !isForwardVisited(myTag) || auxRange[row])
        return;

    uint32_t myRange = range[row];
    uint32_t myPr = Pr[row];
	uint32_t cnt = Fr[row + 1] - Fr[row];
    const uint32_t *nbrs = &Fc[Fr[row]];

	bool end = true;
	for ( uint32_t i = 0; i < cnt; i++ ) {
        uint32_t index = nbrs[i];
        uint8_t nbrTag = tags[index];

		if(isRangeSet(nbrTag) || isForwardVisited(nbrTag) || range[index] != myRange || Pr[index] != myPr || auxRange[index])
			continue;

		setForwardVisitedBit(&tags[index]);
        //printf("Forward visit set for %d from %d\n", index, row);
		end = false;
	}
	setForwardPropagateBit(&tags[row]);
	if (!end)
		*terminate = false;
}


__global__ void bwd(const uint32_t *Bc, const uint32_t *Br, const uint32_t *range, uint8_t *tags, const uint32_t num_rows, bool volatile *terminate){

	uint32_t row = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x + 1;
	uint8_t myTag;

    if (row > num_rows || isRangeSet(myTag = tags[row]) || isBackwardPropagate(myTag) || !isBackwardVisited(myTag))
        return;

    uint32_t myRange = range[row];
	uint32_t cnt = Br[row + 1] - Br[row];
    const uint32_t *nbrs = &Bc[Br[row]];

	bool end = true;
	for ( uint32_t i = 0; i < cnt; i++ ) {
        uint32_t index = nbrs[i];
        uint8_t nbrTag = tags[index];

		if(isRangeSet(nbrTag) || isBackwardVisited(nbrTag) || range[index] != myRange )
			continue;

		setBackwardVisitedBit(&tags[index]);
		end = false;
	}
	setBackwardPropagateBit(&tags[row]);
	if (!end)
		*terminate = false;
}

__global__ void bwdLocal(const uint32_t *Bc, const uint32_t *Br, const uint32_t *range, uint8_t *tags, uint32_t *Pr, const uint32_t num_rows, bool volatile *terminate, bool *auxRange){

	uint32_t row = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x + 1;
	uint8_t myTag;

    if (row > num_rows || isRangeSet(myTag = tags[row]) || isBackwardPropagate(myTag) || !isBackwardVisited(myTag) || auxRange[row])
        return;

    uint32_t myRange = range[row];
    uint32_t myPr = Pr[row];
	uint32_t cnt = Br[row + 1] - Br[row];
    const uint32_t *nbrs = &Bc[Br[row]];

	bool end = true;
	for ( uint32_t i = 0; i < cnt; i++ ) {
        uint32_t index = nbrs[i];
        uint8_t nbrTag = tags[index];

		if(isRangeSet(nbrTag) || isBackwardVisited(nbrTag) || range[index] != myRange || Pr[index] != myPr || auxRange[index])
			continue;

		setBackwardVisitedBit(&tags[index]);
        //printf("Backward visit set for %d from %d\n", index, row);
		end = false;
	}
	setBackwardPropagateBit(&tags[row]);
	if (!end)
		*terminate = false;
}

__global__ void assignUniqueRange(uint32_t *range, const uint8_t *tags, const uint32_t num_rows){
    uint32_t row = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x + 1;

    if (row > num_rows || isRangeSet(tags[row]))
        return;

    range[row] = row;
}

__global__ void assignPartitionRange(uint32_t *range, const uint8_t *tags, uint32_t *Pr, const uint32_t num_rows){
    uint32_t row = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x + 1;

    if (row > num_rows || isRangeSet(tags[row]))
        return;

    range[row] = Pr[row];
}


__global__ void propagateRange1(const uint32_t *Fc, const uint32_t *Fr, uint32_t *range, const uint8_t *tags, const uint32_t num_rows, bool volatile *terminate){
    uint32_t row = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x + 1;

    if (row > num_rows || isRangeSet(tags[row]))
        return;

    uint32_t myRange = range[row];
    uint32_t cnt = Fr[row + 1] - Fr[row];
    const uint32_t *nbrs = &Fc[Fr[row]];
    bool end = true;

    for ( uint32_t i = 0; i < cnt; i++ ) {
        uint32_t index = nbrs[i];
        uint32_t nbrRange = range[index];

        if(!isRangeSet(tags[index]) && nbrRange < myRange){
            myRange = nbrRange;
            end = false;
        }
    }

    if(!end){
        range[row] = myRange;
        *terminate = false;
    }
}

__global__ void propagateRange2(uint32_t *range, const uint8_t *tags, const uint32_t num_rows, bool volatile *terminate){
    uint32_t row = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x + 1;

    if (row > num_rows || isRangeSet(tags[row]))
        return;

    uint32_t myRange = range[row];
    uint32_t newRange;

    if(myRange != row && myRange != (newRange = range[myRange])){
        range[row] = newRange;
        *terminate = false;
    }
}

//Coloring
__global__ void colorPropagation(const uint32_t *Fc, const uint32_t *Fr, uint32_t *range, const uint8_t *tags, const uint32_t num_rows, bool volatile *terminate){
    uint32_t row = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x + 1;

    if (row > num_rows || isRangeSet(tags[row]))
        return;

    uint32_t mx = max(row, range[row]);
    uint32_t cnt = Fr[row + 1] - Fr[row];
    const uint32_t *nbrs = &Fc[Fr[row]];
    bool end = true;

    for ( uint32_t i = 0; i < cnt; i++ ) {
        uint32_t index = nbrs[i];
        uint32_t nbrRange = range[index];

        if(!isRangeSet(tags[index]) && mx < nbrRange){
            mx = nbrRange;
            end = false;
        }
    }

    if(!end){
        range[row] = mx;
        *terminate = false;
    }
}

//coloring
__global__ void selectPivotColoring(const uint32_t *range, uint8_t *tags, const uint32_t num_rows){
    uint32_t row = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x + 1;
    uint8_t myTag;

    if (row > num_rows || isRangeSet(myTag = tags[row]))
        return;

    if(range[row] == row){
        myTag = 0;
        setForwardVisitedBit(&myTag);
        setPivot(&myTag);
        tags[row] = myTag;
    }
}


//coloring
__global__ void fwdColoring(const uint32_t *Fc, const uint32_t *Fr, const uint32_t *range, uint8_t *tags, const uint32_t num_rows, bool volatile *terminate){

    uint32_t row = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x + 1;
    uint8_t myTag;

    if (row > num_rows || isRangeSet(myTag = tags[row]) || !isForwardVisited(myTag))
        return;

    uint32_t myRange = range[row];
    uint32_t cnt = Fr[row + 1] - Fr[row];
    const uint32_t *nbrs = &Fc[Fr[row]];

    bool end = true;
    for ( uint32_t i = 0; i < cnt; i++ ) {
        uint32_t index = nbrs[i];
        uint8_t nbrTag = tags[index];

        if(isRangeSet(nbrTag) || isForwardVisited(nbrTag) || range[index] != myRange)
            continue;

        setForwardVisitedBit(&tags[index]);
        end = false;
    }
    rangeSet(&tags[row]);
    if (!end)
        *terminate = false;
}


//coloring
__global__ void updateColoring(uint8_t *tags, const uint32_t num_rows, bool volatile *terminate){

    uint32_t row = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x + 1;

    if (row > num_rows || isRangeSet(tags[row]))
        return;

    *terminate = false;
    tags[row] = 0;
}


__global__ void selectFirstPivot(uint8_t *tags, const uint32_t num_rows, const uint32_t *pivot_field){

    uint32_t row = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x + 1;
    uint8_t myTag;

    if (row > num_rows || isRangeSet(myTag = tags[row]))
        return;

    if( pivot_field[0] == row ) {
        myTag = 0;
        setForwardVisitedBit(&myTag);
        setBackwardVisitedBit(&myTag);
        setPivot(&myTag);
        tags[row] = myTag;
    }
}

__global__ void pollForFirstPivot(const uint8_t *tags, const uint32_t num_rows, uint32_t* pivot_field, const uint32_t *Fr, const uint32_t *Br){

    uint32_t row = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x + 1;

    if (row > num_rows || isRangeSet(tags[row]))
        return;

    uint32_t oldRow = pivot_field[0];
    uint32_t oldDegree = (Fr[oldRow+1] - Fr[oldRow]) * (Br[oldRow+1] - Br[oldRow]);
    uint32_t newDegree = (Fr[row+1] - Fr[row]) * (Br[row+1] - Br[row]);

    if(newDegree > oldDegree)
        pivot_field[0] = row;
}

/*template<int w>
    int fun(int x){
        int k = w + x;
        return k;
    }*/


__global__ void computeInDegree(const uint8_t *tags, const uint32_t num_rows, uint32_t* Pr, const uint32_t *Br, const uint32_t *Bc, bool *Occ, bool volatile *terminate)
{
    uint32_t row = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x + 1;

    if (row > num_rows || isRangeSet(tags[row]) || Occ[row])
    {
        if(row <= num_rows)
        {
            Occ[row] = true;
        }
        return;
    }

    uint32_t myPr = Pr[row];
    uint32_t frpr;
    if(myPr & 1)
        frpr = myPr - 1;
    else
        frpr = myPr + 1;
    uint32_t cnt = Br[row + 1] - Br[row];
    const uint32_t *nbrs = &Bc[Br[row]];
    uint32_t ideg = 0;

	for ( uint32_t i = 0; i < cnt; i++ ) {
        if(Occ[nbrs[i]] || isRangeSet(tags[nbrs[i]]))
            continue;
        if(Pr[nbrs[i]] == myPr || Pr[nbrs[i]] == frpr)
            ideg++;
	}

	if(ideg == 0)
    {
        Occ[row] = true;
        //printf("Occ true in IN for %d\n", row);
        *terminate = false;
    }
    else
        Occ[row] = false;

}

__global__ void computeOutDegree(const uint8_t *tags, const uint32_t num_rows, uint32_t* Pr, const uint32_t *Fr, const uint32_t *Fc, bool *Occ, bool volatile *terminate)
{
    uint32_t row = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x + 1;

    if (row > num_rows || isRangeSet(tags[row]) || Occ[row])
    {
        if(row <= num_rows)
        {
            Occ[row] = true;
        }
        return;
    }
    uint32_t myPr = Pr[row];
    uint32_t frpr;
    if(myPr & 1)
        frpr = myPr - 1;
    else
        frpr = myPr + 1;
    uint32_t cnt = Fr[row + 1] - Fr[row];
    const uint32_t *nbrs = &Fc[Fr[row]];
    uint32_t odeg = 0;

	for ( uint32_t i = 0; i < cnt; i++ ) {
        if(Occ[nbrs[i]] || isRangeSet(tags[nbrs[i]]))
            continue;
        if(Pr[nbrs[i]] == myPr || Pr[nbrs[i]] == frpr)
            odeg++;
	}

	if(odeg == 0)
    {
        Occ[row] = true;
        //printf("Occ true in OUT for %d\n", row);
        *terminate = false;
    }
    else
        Occ[row] = false;
}

__global__ void getMaxRange(uint32_t *range, uint32_t *Pr, uint32_t *Rm, const uint32_t num_rows, uint8_t *tags, bool volatile *terminate)
{
    uint32_t row = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x + 1;

    if (row > num_rows || isRangeSet(tags[row]))
        return;

    uint32_t myPr = Pr[row];
    if(myPr & 1)
        return;

    //printf("row %d range %d\n", row, range[row]);
    if(Rm[myPr] < range[row])
    {
        Rm[myPr] = range[row];
        //printf("Partition %d Max %d\n", myPr, Rm[myPr]);
        *terminate = false;
    }

}

__global__ void shiftRange(uint32_t *range, uint32_t *Pr, uint32_t *Rm, const uint32_t num_rows, uint8_t *tags)
{
    uint32_t row = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x + 1;

    if (row > num_rows || isRangeSet(tags[row]))
        return;

    uint32_t myPr = Pr[row];
    //printf("GOD: ROW %d range[ROW] %d\n", row, range[row]);
    if(!(myPr & 1))
        return;

    range[row] = range[row] + Rm[myPr - 1] + 1;
    //printf("Shifting parition %d, row range %d by %d\n", myPr, range[row], Rm[myPr - 1] + 1);
}

__global__ void updatePr(uint32_t *Pr, const uint32_t num_rows, bool volatile *terminate, uint8_t *tags)
{
    uint32_t row = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x + 1;
    if (row > num_rows || isRangeSet(tags[row]))
        return;
    uint32_t myPr = Pr[row];
    if(myPr & 1)
    {
        Pr[row] = myPr - 1;
        *terminate = false;
    }
    Pr[row] >>= 1;
    //printf("Updating Pr: %d to %d\n", row, Pr[row]);
}

__global__ void fwdRc(const uint32_t *Fc, const uint32_t *Fr, const uint32_t *range, uint8_t *tags, uint32_t *Pr, const uint32_t num_rows, bool volatile *terminate){

	uint32_t row = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x + 1;
	uint8_t myTag;

    if (row > num_rows || isRangeSet(myTag = tags[row]) || isForwardPropagate(myTag) || !isForwardVisited(myTag))
        return;

    uint32_t myPr = Pr[row];
	uint32_t cnt = Fr[row + 1] - Fr[row];
    const uint32_t *nbrs = &Fc[Fr[row]];

	bool end = true;
	for ( uint32_t i = 0; i < cnt; i++ ) {
        uint32_t index = nbrs[i];
        uint8_t nbrTag = tags[index];
		if(isRangeSet(nbrTag) || isForwardVisited(nbrTag) || Pr[index] != myPr)
			continue;

		setForwardVisitedBit(&tags[index]);
        //printf("Forward visit set for %d from %d\n", index, row);
		end = false;
	}
	setForwardPropagateBit(&tags[row]);
	if (!end)
		*terminate = false;
}

__global__ void bwdRc(const uint32_t *Bc, const uint32_t *Br, const uint32_t *range, uint8_t *tags, uint32_t *Pr, const uint32_t num_rows, bool volatile *terminate){

	uint32_t row = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x + 1;
	uint8_t myTag;

    if (row > num_rows || isRangeSet(myTag = tags[row]) || isBackwardPropagate(myTag) || !isBackwardVisited(myTag))
        return;

    uint32_t myPr = Pr[row];
	uint32_t cnt = Br[row + 1] - Br[row];
    const uint32_t *nbrs = &Bc[Br[row]];

	bool end = true;
	for ( uint32_t i = 0; i < cnt; i++ ) {
        uint32_t index = nbrs[i];
        uint8_t nbrTag = tags[index];

		if(isRangeSet(nbrTag) || isBackwardVisited(nbrTag) || Pr[index] != myPr)
			continue;

		setBackwardVisitedBit(&tags[index]);
        //printf("Backward visit set for %d from %d\n", index, row);
		end = false;
	}
	setBackwardPropagateBit(&tags[row]);
	if (!end)
		*terminate = false;
}
